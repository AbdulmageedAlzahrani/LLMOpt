#include <cstdio>
#include <vector>
#include <hip/hip_runtime.h>

__global__ void vec_add(const float* a, const float* b, float* c, int n){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) c[i] = a[i] + b[i];
}

int main() {
    const int N = 1<<20;
    const size_t bytes = N * sizeof(float);

    std::vector<float> h_a(N, 1.0f), h_b(N, 2.0f), h_c(N);
    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, bytes); hipMalloc(&d_b, bytes); hipMalloc(&d_c, bytes);
    hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b.data(), bytes, hipMemcpyHostToDevice);

    dim3 block(256), grid((N + block.x - 1) / block.x);
    vec_add<<<grid, block>>>(d_a, d_b, d_c, N);
    hipDeviceSynchronize();

    hipMemcpy(h_c.data(), d_c, bytes, hipMemcpyDeviceToHost);
    printf("c[0]=%.1f  c[N-1]=%.1f\n", h_c[0], h_c[N-1]);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    return 0;
}
